#include "hip/hip_runtime.h"

#include "GraphicEngineCUDA.cuh"
#include "__graphic_engine_/GraphicEngine.h"

#include<iostream>


void GraphicEngine::AllocateVertex2D() {

	unsigned int vertexs2d_size = data_info_.numberOfVertexs * sizeof(Vertex2D);

	hipMalloc((void**)& device_vertexs_2d_, vertexs2d_size);
	hipHostMalloc((void**)& host_vertexs_2d_, vertexs2d_size);

};


__global__ void ProjectVertexs(const Vertex3D* const vertexs_3d, Vertex2D* const vertexs_2d, const unsigned int number_of_vertexs, const Vertex3D cameraPosition, const Vector3D vectorX, const Vector3D vectorY, const Vector3D vectorZ, const float k) {

	int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (threadIndex < number_of_vertexs) {

		//Offset

		//Central projection
		float offset_x = vertexs_3d[threadIndex].x - cameraPosition.x;
		float offset_y = vertexs_3d[threadIndex].y - cameraPosition.y;
		float offset_z = vertexs_3d[threadIndex].z - cameraPosition.z;

		float new_x = offset_x * vectorX.x + offset_y * vectorX.y + offset_z * vectorX.z;
		float new_y = offset_x * vectorY.x + offset_y * vectorY.y + offset_z * vectorY.z;
		float new_z = offset_x * vectorZ.x + offset_y * vectorZ.y + offset_z * vectorZ.z;
		
		vertexs_2d[threadIndex].x = ( k * new_x / (new_z + k) ) ;
		vertexs_2d[threadIndex].y = ( k * new_y / (new_z + k) ) ;
		//Ortogonal
		//vertexs_2d[threadIndex].x = vertexs_3d[threadIndex].x;
		//vertexs_2d[threadIndex].y = vertexs_3d[threadIndex].y;

	}

}

__global__ void ConvertInDisplayCoordinats(Vertex2D* const vertexs_2d, const unsigned int number_of_vertexs, const unsigned int display_width, const unsigned int display_height) {
	
	int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;

	if (threadIndex < number_of_vertexs) {

		vertexs_2d[threadIndex].x = vertexs_2d[threadIndex].x * 1000.0f + display_width / 2;
		vertexs_2d[threadIndex].y = display_height / 2 - vertexs_2d[threadIndex].y * 1000.0f;
		
	}

}
__global__ void DrawLines(const Vertex2D* const vertexs_2d, const Polygon3D* const device_polygons, const Normal3D* normals, const unsigned int number_of_polygons, RgbPixel* const display_buffer, const unsigned int display_width, const unsigned int display_height, Vector3D camera_dir) {

	int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;

	if (threadIndex < number_of_polygons * 3) {

		const unsigned int polygon_number = threadIndex / 3;
		const unsigned int vertex_number = threadIndex % 3;
		
		Normal3D normal = normals[ device_polygons[polygon_number].ratios[vertex_number].normalNumber];
		float scalar = camera_dir.x * normal.x + camera_dir.y * normal.y + camera_dir.z * normal.z;
		//if (scalar <= 0.0f) 
		{

			const unsigned int first_vertex_index = device_polygons[polygon_number].ratios[vertex_number].vertexNumber;
			const unsigned int second_vertex_number = ((vertex_number + 1) < 3) ? (vertex_number + 1) : 0;
			const unsigned int second_vertex_index = device_polygons[polygon_number].ratios[second_vertex_number].vertexNumber;

			int x1 = vertexs_2d[first_vertex_index].x;
			int y1 = vertexs_2d[first_vertex_index].y;
			const int x2 = vertexs_2d[second_vertex_index].x;
			const int y2 = vertexs_2d[second_vertex_index].y;

			const bool coordinats_are_correct = (x1 > 0 && x1 < display_width) && (x2 > 0 && x2 < display_width) && (y1 > 0 && y1 < display_height) && (y2 > 0 && y2 < display_height);

			if (coordinats_are_correct) {

				const int deltaX = abs(x2 - x1);
				const int deltaY = abs(y2 - y1);
				const int signX = x1 < x2 ? 1 : -1;
				const int signY = y1 < y2 ? 1 : -1;
				//
				int error = deltaX - deltaY;

				display_buffer[display_width * y2 + x2].rgb_reserved = 0;
				display_buffer[display_width * y2 + x2].rgb_red = 255;
				display_buffer[display_width * y2 + x2].rgb_green = 0;
				display_buffer[display_width * y2 + x2].rgb_blue = 0;

				while (x1 != x2 || y1 != y2)
				{

					display_buffer[display_width * y1 + x1].rgb_reserved = 0;
					display_buffer[display_width * y1 + x1].rgb_red = 255;
					display_buffer[display_width * y1 + x1].rgb_green = 0;
					display_buffer[display_width * y1 + x1].rgb_blue = 0;

					const int error2 = error * 2;
					//
					if (error2 > -deltaY)
					{
						error -= deltaY;
						x1 += signX;
					}

					if (error2 < deltaX)
					{
						error += deltaX;
						y1 += signY;
					}

				}
			}
		}
	}

}
__global__ void  SetScreenColor(RgbPixel* const device_display_buffer, const RgbPixel rgb_pixel, const unsigned int number_of_pixels) {

	int thread_index = threadIdx.x + blockIdx.x * blockDim.x;

	if (thread_index < number_of_pixels) device_display_buffer[thread_index] = rgb_pixel;

}

void GraphicEngine::SetDisplayBufferColor(const RgbColor& rgb_color)
{
	const RgbPixel rgb_pixel = { rgb_color.rgb_blue, rgb_color.rgb_green, rgb_color.rgb_red, rgb_color.rgb_reserved };

	const unsigned int number_of_threads = 1024;

	const unsigned int number_of_blocks = (1920 * 1080 * sizeof(RgbPixel) + number_of_threads - 1) / number_of_threads;

	SetScreenColor <<< number_of_blocks, number_of_threads >>> (this->device_display_buffer_, rgb_pixel, 1920 * 1080);


}
void CPUCountingProjectVertexs(const Vertex3D* const vertexs_3d, Vertex2D* const vertexs_2d, const unsigned int number_of_vertexs, const Vertex3D cameraPosition, const Vector3D vectorX, const Vector3D vectorY, const Vector3D vectorZ, const float k) {

	

	for (size_t i = 0; i < number_of_vertexs; i++)
	{



		//Offset
		const float offset_x = vertexs_3d[i].x - cameraPosition.x;
		const float offset_y = vertexs_3d[i].y - cameraPosition.y;
		const float offset_z = vertexs_3d[i].z - cameraPosition.z;

		const float new_x = offset_x * vectorX.x + offset_y * vectorX.y + offset_z * vectorX.z;
		const float new_y = offset_x * vectorY.x + offset_y * vectorY.y + offset_z * vectorY.z;
		const float new_z = offset_x * vectorZ.x + offset_y * vectorZ.y + offset_z * vectorZ.z;

		vertexs_2d[i].x = (k * new_x / (new_z + k));
		vertexs_2d[i].y = (k * new_y / (new_z + k));
	}
	
}
void CPUCountingConvertInDisplayCoordinats(Vertex2D* const vertexs_2d, const unsigned int number_of_vertexs, const unsigned int display_width, const unsigned int display_height) {

	

	for (size_t i = 0; i < number_of_vertexs; i++)
	{

		vertexs_2d[i].x = vertexs_2d[i].x * (float) 1000 + display_width / 2;
		vertexs_2d[i].y = display_height / 2 - vertexs_2d[i].y * 1000;

	}
}
void CPUCountingDrawLines(const Vertex2D* const vertexs_2d, const Polygon3D* const device_polygons, const unsigned int number_of_polygons, RgbPixel* const display_buffer, const unsigned int display_width, const unsigned int display_height) {

	{
		const unsigned int number_of_lines = number_of_polygons * 3;

		for (size_t i = 0; i < number_of_lines; i++)
		{

			const unsigned int polygon_number = i / 3;
			const unsigned int vertex_number = i % 3;

			const unsigned int first_vertex_index = device_polygons[polygon_number].ratios[vertex_number].vertexNumber;
			const unsigned int second_vertex_number = ((vertex_number + 1) < 3) ? (vertex_number + 1) : 0;
			const unsigned int second_vertex_index = device_polygons[polygon_number].ratios[second_vertex_number].vertexNumber;


			int x1 = vertexs_2d[first_vertex_index].x;
			int y1 = vertexs_2d[first_vertex_index].y;
			int x2 = vertexs_2d[second_vertex_index].x;
			int y2 = vertexs_2d[second_vertex_index].y;

			const bool coordinats_are_correct = (x1 > 0 && x1 < display_width) && (x2 > 0 && x2 < display_width) && (y1 > 0 && y1 < display_height) && (y2 > 0 && y2 < display_height);

			if (coordinats_are_correct) {

				const int deltaX = abs(x2 - x1);
				const int deltaY = abs(y2 - y1);
				const int signX = x1 < x2 ? 1 : -1;
				const int signY = y1 < y2 ? 1 : -1;
				//
				int error = deltaX - deltaY;

				display_buffer[display_width * y2 + x2].rgb_reserved = 0;
				display_buffer[display_width * y2 + x2].rgb_red = 255;
				display_buffer[display_width * y2 + x2].rgb_green = 0;
				display_buffer[display_width * y2 + x2].rgb_blue = 0;

				/*display_buffer[display_width * y1 + x1].rgb_reserved = 0;
				display_buffer[display_width * y1 + x1].rgb_red = 255;
				display_buffer[display_width * y1 + x1].rgb_green = 0;
				display_buffer[display_width * y1 + x1].rgb_blue = 0;*/


				while (x1 != x2 || y1 != y2)
				{
					display_buffer[display_width * y1 + x1].rgb_reserved = 0;
					display_buffer[display_width * y1 + x1].rgb_red = 255;
					display_buffer[display_width * y1 + x1].rgb_green = 0;
					display_buffer[display_width * y1 + x1].rgb_blue = 0;

					const int error2 = error * 2;
					//
					if (error2 > -deltaY)
					{
						error -= deltaY;
						x1 += signX;
					}
					if (error2 < deltaX)
					{
						error += deltaX;
						y1 += signY;
					}
				}

			}
		}
	}
}
__global__ void draw(RgbPixel* display_buffer) {
	int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

	display_buffer[thread_id].rgb_green = 255;

}

CameraInfo GraphicEngine::GetCameraInfo() {
	CameraInfo info;
	info.camera_pos = (*camera_->GetPosition());
	info.dis_proj_plane = camera_->GetDistanceToProjPlane();
	info.vector_x = camera_->GetVectorX();
	info.vector_y = camera_->GetVectorY();
	info.vector_z = camera_->GetDirection();
	return info;
}
void GraphicEngine::TestFunction()
{


}

void GraphicEngine::CreateMeshFrame() {

	const Vertex3D const  camera_position  = *(camera_->GetPosition());
	
	const Vector3D vector_x = camera_->GetVectorX();
	const Vector3D vector_y = camera_->GetVectorY();
	const Vector3D vector_z = camera_->GetDirection();

	const float distance_to_projection_plane = camera_->GetDistanceToProjPlane();

	const Vertex3D* const device_vertexs_3d = device_data_.deviceVertexs;
	Vertex2D* const device_vertexs_2d = device_vertexs_2d_;
	const Polygon3D* const device_polygons = device_data_.devicePolygons;
	Normal3D* device_normals = device_data_.deviceNormals;
	
	RgbColor color;
	color.rgb_blue = 20;
	color.rgb_green = 255;
	color.rgb_red = 0;

	const unsigned int number_of_threads = 1024;
	unsigned int number_of_blocks = (data_info_.numberOfVertexs + number_of_threads - 1) / number_of_threads;

	SetDisplayBufferColor(color);

	ProjectVertexs <<<number_of_blocks, number_of_threads>>> (device_vertexs_3d, device_vertexs_2d, data_info_.numberOfVertexs, camera_position, vector_x, vector_y, vector_z, distance_to_projection_plane);

	ConvertInDisplayCoordinats <<<number_of_blocks, number_of_threads >>> (device_vertexs_2d, data_info_.numberOfVertexs, display_width_, display_height_);

	number_of_blocks = (data_info_.numberOfPolygons * 3 + number_of_threads - 1) / number_of_threads;

	DrawLines <<<number_of_blocks, number_of_threads >>> (device_vertexs_2d, device_polygons, device_normals, data_info_.numberOfPolygons, device_display_buffer_, display_width_, display_height_,vector_z);	

	//hipMemcpy((void**)host_display_buffer_, device_display_buffer_, display_buffer_size_, hipMemcpyDeviceToHost);
	//const Vertex3D* const host_vertexs_3d = data_info_.allVertexs;
	//hipMemcpy(host_display_buffer_, device_display_buffer_, size_of_display_buffer_, hipMemcpyDeviceToHost);
	//CPUCountingProjectVertexs(host_vertexs_3d, this->host_vertexs_2d_, this->data_info_.numberOfVertexs, camera_position, vector_x, vector_y, vector_z, distance_to_projection_plane);
	//CPUCountingConvertInDisplayCoordinats(host_vertexs_2d_, this->data_info_.numberOfVertexs, display_width_, display_height_);
	//Polygon3D* host_polygons = this->data_info_.allPolygons;
	//CPUCountingDrawLines(host_vertexs_2d_, host_polygons, this->data_info_.numberOfPolygons, this->host_display_buffer_, display_width_, display_height_);
	//hipMemcpy(host_display_buffer_, device_display_buffer_, size_of_display_buffer_, hipMemcpyDeviceToHost);

}


struct Proj_vertex {

	float x;
	float y;
	float _z;

};
inline __device__ void swap(Proj_vertex& a, Proj_vertex& b) {

	Proj_vertex temporary = b;
	b = a;
	a = temporary;

}
inline __device__ bool InPositiveHalfPlane(const Vertex2D& pixel, const Vertex2D& triangle_vertex, Vector2D& _normal) {

	Vector2D pixel_vector;
	pixel_vector.x = pixel.x - triangle_vertex.x;
	pixel_vector.y = pixel.y - triangle_vertex.y;

	Vector2D normal = _normal;
	float length_n = sqrt(normal.x * normal.x + normal.y * normal.y);
	normal.x /= length_n;
	normal.y /= length_n;

	float length_p = sqrt(pixel_vector.x * pixel_vector.x +  pixel_vector.y * pixel_vector.y);
	pixel_vector.x /= length_p;
	pixel_vector.y /=length_p;
	float scalar = pixel_vector.x * normal.x + pixel_vector.y * normal.y;

	if (scalar > 0.0f) return true;
	else 
		return false;

}

struct InfoForPainting {
	Vertex3D* d_vertexs;
	Polygon3D* d_polygons;
	unsigned int number_of_polygons;
	//RgbPixel* d_rgb;
	//unsigned int number_of_colors;
	unsigned int threads_per_triangle;
	unsigned int screen_width;
	unsigned int screen_height;
	Vertex3D camera_position;
	Vector3D vector_x;
	Vector3D vector_y;

};


struct Device_vertex3D{
	float x;
	float y;
	float z;
};

struct InEl {

	int x;
	int y;
	float w;

};

__device__ float Dist_plane_vertex(const  Device_vertex3D& plane_v_1, const  Device_vertex3D& plane_v_2, const  Device_vertex3D& plane_v_3, const  Device_vertex3D& vertex) {

	float A, B, C, D;
	A = plane_v_1.y*(plane_v_2.z - plane_v_3.z) + plane_v_2.y*(plane_v_3.z - plane_v_1.z) + plane_v_3.y*(plane_v_1.z - plane_v_2.z);
	B = plane_v_1.z*(plane_v_2.x - plane_v_3.x) + plane_v_2.z*(plane_v_3.x - plane_v_1.x) + plane_v_3.z*(plane_v_1.x - plane_v_2.x);
	C = plane_v_1.x*(plane_v_2.y - plane_v_3.y) + plane_v_2.x*(plane_v_3.y - plane_v_1.y) + plane_v_3.x*(plane_v_1.y - plane_v_2.y);
	D = plane_v_1.x*(plane_v_2.y* plane_v_3.z - plane_v_3.y* plane_v_2.z) + plane_v_2.x*(plane_v_3.y* plane_v_1.z - plane_v_1.y* plane_v_3.z) + plane_v_3.x*(plane_v_1.y* plane_v_2.z - plane_v_2.y* plane_v_1.z);
	D = -D;
	float numerator = (A * vertex.x + B * vertex.y + C * vertex.y + D);
	if (numerator < 0)numerator = -numerator;
	return numerator / sqrt(A*A + B*B + C*C);

}
__device__ float Interpolate(float y1, float I1, float y2, float I2, float ya) {

	
	float Ip = I1 * ((ya - y2) / (y1 - y2)) + I2 * ((y1 - ya) / (y1 - y2));
	
	return Ip;

}

__device__ float BiInterpolate(InEl el_1, InEl el_2, InEl el_3, int x, int y) {

	//float fxy1 = 

}

__global__ void DrawPolygons(int* mutex_buffer, float* w_buffer, RgbPixel* display_buffer, Vertex2D* vertexs_2d, InfoForPainting info) {

	int thread_index = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (thread_index < info.number_of_polygons * info.threads_per_triangle) {
		
		const unsigned int thread_index_in_triangle = (thread_index % info.threads_per_triangle);

		Polygon3D polygon = info.d_polygons[thread_index / info.threads_per_triangle];

		Proj_vertex proj_vertexs[3];

		for (int i = 0; i < 3; i++)
		{

			proj_vertexs[i].x = vertexs_2d[polygon.ratios[i].vertexNumber].x;
			//printf("x: %2f", proj_vertexs[i].x);
			proj_vertexs[i].y = vertexs_2d[polygon.ratios[i].vertexNumber].y;
			//printf("y: %2f \n", proj_vertexs[i].y);

			//printf("%f %f %f \n %f %f %f \n", info.d_vertexs[polygon.ratios[i].vertexNumber].x, info.d_vertexs[polygon.ratios[i].vertexNumber].y, info.d_vertexs[polygon.ratios[i].vertexNumber].z, info.camera_position.x, info.camera_position.y, info.camera_position.z);

			/*float x_2 = info.d_vertexs[polygon.ratios[i].vertexNumber].x - info.camera_position.x;
			float y_2 = info.d_vertexs[polygon.ratios[i].vertexNumber].y - info.camera_position.y;
			float z_2 = info.d_vertexs[polygon.ratios[i].vertexNumber].z - info.camera_position.z;
			float distance_to_camera = sqrt(x_2 * x_2 + y_2 * y_2 +z_2 * z_2 );*/
			//printf("%f \n", distance_to_camera);

			Device_vertex3D plane_v_1, plane_v_2, plane_v_3, vertex;

				plane_v_1.x = info.camera_position.x;
				plane_v_1.y = info.camera_position.y;
				plane_v_1.z = info.camera_position.z;

				plane_v_2.x = info.vector_x.x;
				plane_v_2.y = info.vector_x.y;
				plane_v_2.z = info.vector_x.z;

				plane_v_3.x = info.vector_y.x;
				plane_v_3.y = info.vector_y.y;
				plane_v_3.z = info.vector_y.z;

				vertex.x = info.d_vertexs[polygon.ratios[i].vertexNumber].x;
				vertex.y = info.d_vertexs[polygon.ratios[i].vertexNumber].y;
				vertex.z = info.d_vertexs[polygon.ratios[i].vertexNumber].z;
				
				float distance_to_camera = Dist_plane_vertex(plane_v_1, plane_v_2, plane_v_3, vertex);
				proj_vertexs[i]._z = 1.0f / distance_to_camera;

			}


			RgbPixel polygon_color = polygon.color;

			//Clockwise direction
			Vertex2D AToB;
			AToB.x = proj_vertexs[1].x - proj_vertexs[0].x;
			AToB.y = proj_vertexs[1].y - proj_vertexs[0].y;
			Vertex2D BToC;
			BToC.x = proj_vertexs[2].x - proj_vertexs[1].x;
			BToC.y = proj_vertexs[2].y - proj_vertexs[1].y;

			float crossz = AToB.x * BToC.y - AToB.y * BToC.x;
			if (crossz > 0.0f)
			{
				Proj_vertex temporary = proj_vertexs[2];
				proj_vertexs[2] = proj_vertexs[1];
				proj_vertexs[1] = temporary;
			}

			//Normalize 2d vectors
			float length;

			Vector2D bot_mid = { proj_vertexs[1].y - proj_vertexs[0].y, -proj_vertexs[1].x + proj_vertexs[0].x };
			length = sqrt(bot_mid.x * bot_mid.x + bot_mid.y * bot_mid.y);
			bot_mid.x /= length;
			bot_mid.y /= length;

			Vector2D mid_top = { proj_vertexs[2].y - proj_vertexs[1].y,  -proj_vertexs[2].x + proj_vertexs[1].x };
			length = sqrt(mid_top.x * mid_top.x + mid_top.y * mid_top.y);
			mid_top.x /= length;
			mid_top.y /= length;

			Vector2D top_bot = { proj_vertexs[0].y - proj_vertexs[2].y, -proj_vertexs[0].x + proj_vertexs[2].x, };
			length = sqrt(top_bot.x * top_bot.x + top_bot.y * top_bot.y);
			top_bot.x /= length;
			top_bot.y /= length;

			const Vertex2D bot = { proj_vertexs[0].x, proj_vertexs[0].y };
			const Vertex2D mid = { proj_vertexs[1].x, proj_vertexs[1].y };
			const Vertex2D top = { proj_vertexs[2].x, proj_vertexs[2].y };

		int min_x = 10000, min_y = 10000, max_x = -1, max_y = -1;

		for (int i = 0; i < 3; i++)
		{
			if (proj_vertexs[i].x < min_x) min_x = floor(proj_vertexs[i].x);
			if (proj_vertexs[i].y < min_y) min_y = floor(proj_vertexs[i].y);
			if (proj_vertexs[i].x > max_x) max_x = ceil(proj_vertexs[i].x);
			if (proj_vertexs[i].y > max_y) max_y = ceil(proj_vertexs[i].y);
		}

		unsigned int delta_y = max_y - min_y;
		unsigned int delta_x = max_x - min_x;
		unsigned int index = thread_index_in_triangle;

		for (int i = 0; i < (delta_y * delta_x) / info.threads_per_triangle; i++)
		{
			unsigned int x = (index % delta_x) + min_x;
			unsigned int y = (index / delta_x) + min_y;

			Vertex2D pixel;
			pixel.x = ((float)x + 0.5f);
			pixel.y = ((float)y + 0.5f);

			bool PixelInTriangle = InPositiveHalfPlane(pixel, bot, bot_mid) && InPositiveHalfPlane(pixel, mid, mid_top) && InPositiveHalfPlane(pixel, top, top_bot);

			if (PixelInTriangle) {
				
				Proj_vertex v[3];

				for (int i = 0; i < 3; i++) v[i] = proj_vertexs[i];

				if (v[0].y < v[1].y) swap(v[0], v[1]);
				if (v[1].y < v[2].y) swap(v[1], v[2]); 
				if (v[0].y < v[1].y) swap(v[0], v[1]);
			

				float I1 = v[0]._z, I2 = v[1]._z, I3 = v[2]._z;

				float X1 = v[0].x, X2 = v[1].x, X3 = v[2].x;


				float Xa, Xb;
				float Ia, Ib, Ip;
				
				if (pixel.y > v[1].y) {

					Ia = Interpolate(v[0].y, I1, v[1].y, I2, pixel.y);
					Xa = Interpolate(v[0].y, v[0].x, v[1].y, v[1].x, pixel.y);

				}
				else {

					Ia = Interpolate(v[2].y, I3, v[1].y, I2, pixel.y);
					Xa = Interpolate(v[1].y, v[1].x, v[2].y, v[2].x, pixel.y);

				}
				
				Ib = Interpolate(v[0].y, I1, v[2].y, I3, pixel.y);
				Xb = Interpolate(v[0].y, v[0].x, v[2].y, v[2].x, pixel.y);
				
				Ip = Interpolate(Xa, Ia, Xb, Ib, pixel.x);


				bool is_set = 0;


				do
				{	
					
					is_set = atomicCAS((mutex_buffer + 1920 * y + x), 0, 1);
					
					if (is_set)
					{

						if (Ip > *(w_buffer + y * 1920 + x)) {

							RgbPixel grey;
							grey.rgb_blue = (int)100.0f * Ip;
							grey.rgb_green = (int)100.0f * Ip;
							grey.rgb_red = (int)100.0f * Ip;
							*(display_buffer + y * 1920 + x) = grey;


						}
						
					}

					if (is_set)
					{

						//atomicExch(mutex_buffer + 1920 * y + x, 0);

						*(mutex_buffer + 1920 * y + x) = 0;
					}

				} while (!is_set);
				
			}

		index += info.threads_per_triangle;
		}
		
	}

	//__syncthreads();
}
__global__ void MemSet(float* memory, UINT el_size, UINT el_num, float value ) {

	int thread_index = threadIdx.x + blockDim.x * blockIdx.x;

	if (thread_index < el_num) {
		*((float*)memory + el_size * thread_index) = value;
		
	}


}
void GraphicEngine::CreateFlatFrame() {

	const Vertex3D const  camera_position = *(camera_->GetPosition());

	const Vector3D vector_x = camera_->GetVectorX();
	const Vector3D vector_y = camera_->GetVectorY();
	const Vector3D vector_z = camera_->GetDirection();

	const float distance_to_projection_plane = camera_->GetDistanceToProjPlane();

	Vertex3D* const device_vertexs_3d = device_data_.deviceVertexs;
	Vertex2D* const device_vertexs_2d = device_vertexs_2d_;
	Polygon3D* const device_polygons = device_data_.devicePolygons;

	RgbColor color;
	color.rgb_blue = 0;
	color.rgb_green = 0;
	color.rgb_red = 0;

	hipMemset(z_mutex_, 0.0f, display_width_ * display_height_ * sizeof(int));
	hipMemset(w_buffer_, 0.0, display_width_ * display_height_ * sizeof(float));

	const unsigned int number_of_threads = 1024;
	//unsigned int number_of_blocks = (display_width_ * display_height_ + number_of_threads - 1) / number_of_threads;;
	
	//MemSet <<<number_of_blocks, number_of_threads >>> (w_buffer_, sizeof(float), display_width_ * display_height_, 0.0f);
	unsigned int number_of_blocks = (data_info_.numberOfVertexs + number_of_threads - 1) / number_of_threads;

	SetDisplayBufferColor(color);

	ProjectVertexs <<< number_of_blocks, number_of_threads >>> (device_vertexs_3d, device_vertexs_2d, data_info_.numberOfVertexs, camera_position, vector_x, vector_y, vector_z, distance_to_projection_plane);

	ConvertInDisplayCoordinats <<< number_of_blocks, number_of_threads >>> (device_vertexs_2d, data_info_.numberOfVertexs, display_width_, display_height_);

	number_of_blocks = (data_info_.numberOfPolygons * threads_per_triangle_ + number_of_threads - 1) / number_of_threads;

	InfoForPainting info = { device_vertexs_3d, device_polygons, data_info_.numberOfPolygons, threads_per_triangle_, display_width_, display_height_, camera_position, vector_x, vector_y };
	//test <<<1, 1 >>> (z_mutex_);
	DrawPolygons <<< number_of_blocks, number_of_threads >>> (z_mutex_, w_buffer_, device_display_buffer_, device_vertexs_2d, info);

	//system("pause");

}



