#include "../../__engine_1.1/__gpu_manager_/CUDAGpuManager.cuh"


void* AllocateMemory(const long long n_bytes) noexcept(true)
{

	void* ptr = nullptr;
	hipMalloc((void**)&ptr, n_bytes);
	return ptr;

}

void MemoryCopy(const void* host_data, void* device_ptr, const long long size)noexcept(true) {

	hipMemcpy((void*)device_ptr, (const void*)host_data, size, hipMemcpyHostToDevice);

}

void FreeMemory(void* device_data)noexcept(true) {

	hipFree(static_cast<void*>(device_data));

}