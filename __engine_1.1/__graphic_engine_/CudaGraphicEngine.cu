#include "hip/hip_runtime.h"
#include "CudaGraphicEngine.cuh"
#include "hip/hip_runtime.h"
#include ""

__global__ void ConvertInDisplayCoordinats(Vertex2D* const vertexs_2d, const unsigned int number_of_vertexs, const unsigned int display_width, const unsigned int display_height) {

	int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;

	if (threadIndex < number_of_vertexs) {

		vertexs_2d[threadIndex].x = vertexs_2d[threadIndex].x * 1000.0f + display_width / 2;
		vertexs_2d[threadIndex].y = display_height / 2 - vertexs_2d[threadIndex].y * 1000.0f;

	}

}

__global__ void DrawLines(const Vertex2D* const vertexs_2d, const Polygon3D* const device_polygons, const Normal3D* normals, const unsigned int number_of_polygons, FramePixel* const display_buffer, const unsigned int display_width, const unsigned int display_height, Vector3D camera_dir) {

	int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;

	if (threadIndex < number_of_polygons * 3) {

		const unsigned int polygon_number = threadIndex / 3;
		const unsigned int vertex_number = threadIndex % 3;

		Normal3D normal = normals[device_polygons[polygon_number].ratios[vertex_number].normal_n];
		float scalar = camera_dir.x * normal.x + camera_dir.y * normal.y + camera_dir.z * normal.z;
		//if (scalar <= 0.0f) 
		{

			const unsigned int first_vertex_index = device_polygons[polygon_number].ratios[vertex_number].vertex_n;
			const unsigned int second_vertex_number = ((vertex_number + 1) < 3) ? (vertex_number + 1) : 0;
			const unsigned int second_vertex_index = device_polygons[polygon_number].ratios[second_vertex_number].vertex_n;

			int x1 = vertexs_2d[first_vertex_index].x;
			int y1 = vertexs_2d[first_vertex_index].y;
			const int x2 = vertexs_2d[second_vertex_index].x;
			const int y2 = vertexs_2d[second_vertex_index].y;

			const bool coordinats_are_correct = (x1 > 0 && x1 < display_width) && (x2 > 0 && x2 < display_width) && (y1 > 0 && y1 < display_height) && (y2 > 0 && y2 < display_height);

			if (coordinats_are_correct) {

				const int deltaX = abs(x2 - x1);
				const int deltaY = abs(y2 - y1);
				const int signX = x1 < x2 ? 1 : -1;
				const int signY = y1 < y2 ? 1 : -1;
				//
				int error = deltaX - deltaY;

				display_buffer[display_width * y2 + x2].rgba_alpha = 0;
				display_buffer[display_width * y2 + x2].rgba_red = 255;
				display_buffer[display_width * y2 + x2].rgba_green = 0;
				display_buffer[display_width * y2 + x2].rgba_blue = 0;

				while (x1 != x2 || y1 != y2)
				{

					display_buffer[display_width * y1 + x1].rgba_alpha = 0;
					display_buffer[display_width * y1 + x1].rgba_red = 255;
					display_buffer[display_width * y1 + x1].rgba_green = 0;
					display_buffer[display_width * y1 + x1].rgba_blue = 0;

					const int error2 = error * 2;
					//
					if (error2 > -deltaY)
					{
						error -= deltaY;
						x1 += signX;
					}

					if (error2 < deltaX)
					{
						error += deltaX;
						y1 += signY;
					}

				}
			}
		}
	}

}

__global__ void ProjectVertexs(const Vertex3D* const vertexs_3d, Vertex2D* const vertexs_2d, const unsigned int number_of_vertexs, const Vertex3D cameraPosition, const Vector3D vectorX, const Vector3D vectorY, const Vector3D vectorZ, const float k) {

	int thread_index = threadIdx.x + blockIdx.x * blockDim.x;

	if (thread_index < number_of_vertexs) {

		//Offset

		//Central projection
		float offset_x = vertexs_3d[thread_index].x - cameraPosition.x;
		float offset_y = vertexs_3d[thread_index].y - cameraPosition.y;
		float offset_z = vertexs_3d[thread_index].z - cameraPosition.z;

		float new_x = offset_x * vectorX.x + offset_y * vectorX.y + offset_z * vectorX.z;
		float new_y = offset_x * vectorY.x + offset_y * vectorY.y + offset_z * vectorY.z;
		float new_z = offset_x * vectorZ.x + offset_y * vectorZ.y + offset_z * vectorZ.z;

		vertexs_2d[thread_index].x = (k * new_x / (new_z + k));
		vertexs_2d[thread_index].y = (k * new_y / (new_z + k));
		//Ortogonal
		//vertexs_2d[threadIndex].x = vertexs_3d[threadIndex].x;
		//vertexs_2d[threadIndex].y = vertexs_3d[threadIndex].y;

	}
}

__global__ void  SetScreenColor(FramePixel* const device_display_buffer, const FramePixel rgb_pixel, const unsigned int number_of_pixels) {

	int thread_index = threadIdx.x + blockIdx.x * blockDim.x;

	if (thread_index < number_of_pixels) device_display_buffer[thread_index] = rgb_pixel;

}

__host__ void GraphicEngine::FillDeviceFrame(const FramePixel& pixel)noexcept(true) {

	const size_t frame_size = device_frame_.Width() * device_frame_.Height();
	const size_t number_of_threads = 1024;
	const size_t number_of_blocks = (frame_size + number_of_threads - 1) / number_of_threads;

	SetScreenColor << < number_of_blocks, number_of_threads >> > (device_frame_.FramePixelPtr(), pixel, frame_size);

}
__host__ void CopyDeviceFrameToHostFrame(FrameHost& host_frame, FrameDevice& device_frame)noexcept(true) {

	hipMemcpy((void*)host_frame.buffer_, (const void*) device_frame.buffer_, host_frame.Width() * host_frame.Height() * sizeof(FramePixel), hipMemcpyDeviceToHost);

}

__host__ FrameHost& GraphicEngine::RenderMeshFrame()noexcept(true) {

	FillDeviceFrame(RgbColor{ 255, 166, 128 });
	/*or (size_t i = 0; i < host_frame_.Width(); i++)
		for (size_t j = 0; j < host_frame_.Height(); j++)
			host_frame_[{i, j}] = color;*/



	static Vertex2D* const device_vertexs_2d = static_cast<Vertex2D* const>(AllocateMemory(device_data_->PrimitivesCapacity().n_vertexs * sizeof(Vertex2D)));

	const Vertex3D& camera_position = camera_->GetPosition();
	//
	const Vector3D& vector_x = camera_->GetVectorX();
	const Vector3D& vector_y = camera_->GetVectorY();
	const Vector3D& vector_z = camera_->GetDirection();

	const float proj_plane_dist = camera_->GetDistanceToProjPlane();

	DevicePtrs ptrs;
	ptrs = device_data_->Ptrs();

	NPrimitives n_primitives;
	n_primitives = device_data_->PrimitivesCapacity();

	const size_t number_of_threads = 1024;
	size_t number_of_blocks = (n_primitives.n_vertexs + number_of_threads - 1) / number_of_threads;

	ProjectVertexs <<<number_of_blocks, number_of_threads>>> (ptrs.vertexs, device_vertexs_2d, n_primitives.n_vertexs, camera_position, vector_x, vector_y, vector_z, proj_plane_dist);

	ConvertInDisplayCoordinats <<<number_of_blocks, number_of_threads >>> (device_vertexs_2d, n_primitives.n_vertexs, device_frame_.Width(), device_frame_.Height());

	number_of_blocks = (n_primitives.n_polygons* 3 + number_of_threads - 1) / number_of_threads;

	DrawLines <<< number_of_blocks, number_of_threads >>> (device_vertexs_2d, ptrs.polygons, ptrs.normals, n_primitives.n_polygons , static_cast<FramePixel*>(static_cast<void*>(device_frame_)), device_frame_.Width(), device_frame_.Height(), vector_z);
	
	CopyDeviceFrameToHostFrame(host_frame_, device_frame_);
	
	return host_frame_;
}
